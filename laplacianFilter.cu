#include "hip/hip_runtime.h"
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

__global__ void laplacianFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float kernel[3][3] = { 0, -1, 0, -1, 4, -1, 0, -1, 0 };

    if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
    {
        float sum = 0;
        for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
            for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
                float fl = srcImage[((y + ky) * width + (x + kx))];
                sum += fl * kernel[ky + FILTER_HEIGHT / 2][kx + FILTER_WIDTH / 2];
            }
        }
        dstImage[(y * width + x)] = sum;
    }
}

extern "C" void laplacianFilter_GPU_wrapper(const cv::Mat & input, cv::Mat & output)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int inputSize = input.cols * input.rows;
    const int outputSize = output.cols * output.rows;
    unsigned char* d_input, * d_output;

    hipMalloc<unsigned char>(&d_input, inputSize);
    hipMalloc<unsigned char>(&d_output, outputSize);

    hipMemcpy(d_input, input.ptr(), inputSize, hipMemcpyHostToDevice);

    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    hipEventRecord(start);

    laplacianFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);

    hipEventRecord(stop);

    hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    hipEventSynchronize(stop);
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "\nProcessing time for GPU (ms): " << milliseconds << "\n";
}