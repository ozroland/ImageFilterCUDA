#include "hip/hip_runtime.h"
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

__global__ void sobelFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float Kx[3][3] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
    float Ky[3][3] = { 1, 2, 1, 0, 0, 0, -1, -2, -1 };

    if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
    {
        float Gx = 0;
        for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
            for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
                float fl = srcImage[((y + ky) * width + (x + kx))];
                Gx += fl * Kx[ky + FILTER_HEIGHT / 2][kx + FILTER_WIDTH / 2];
            }
        }
        float Gx_abs = Gx < 0 ? -Gx : Gx;

        float Gy = 0;
        for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
            for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
                float fl = srcImage[((y + ky) * width + (x + kx))];
                Gy += fl * Ky[ky + FILTER_HEIGHT / 2][kx + FILTER_WIDTH / 2];
            }
        }
        float Gy_abs = Gy < 0 ? -Gy : Gy;

        dstImage[(y * width + x)] = Gx_abs + Gy_abs;
    }
}

extern "C" void sobelFilter_GPU_wrapper(const cv::Mat & input, cv::Mat & output)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int inputSize = input.cols * input.rows;
    const int outputSize = output.cols * output.rows;
    unsigned char* d_input, * d_output;

    hipMalloc<unsigned char>(&d_input, inputSize);
    hipMalloc<unsigned char>(&d_output, outputSize);

    hipMemcpy(d_input, input.ptr(), inputSize, hipMemcpyHostToDevice);

    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    hipEventRecord(start);

    sobelFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);

    hipEventRecord(stop);

    hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    hipEventSynchronize(stop);
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "\nProcessing time on GPU (ms): " << milliseconds << "\n";
}