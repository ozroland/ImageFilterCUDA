#include "hip/hip_runtime.h"
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

__global__ void tvFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
    {
        float sod = 0;
        for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
            for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
                float fl = srcImage[((y + ky) * width + (x + kx))];
                float center = srcImage[((y)*width + (x))];
                sod += fl - center;
            }
        }
        dstImage[(y * width + x)] = sod;
    }
}

extern "C" void tvFilter_GPU_wrapper(const cv::Mat & input, cv::Mat & output)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int inputSize = input.cols * input.rows;
    const int outputSize = output.cols * output.rows;
    unsigned char* d_input, * d_output;

    hipMalloc<unsigned char>(&d_input, inputSize);
    hipMalloc<unsigned char>(&d_output, outputSize);

    hipMemcpy(d_input, input.ptr(), inputSize, hipMemcpyHostToDevice);

    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    hipEventRecord(start);

    tvFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);

    hipEventRecord(stop);

    hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    hipEventSynchronize(stop);
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "\nTotal processing time on GPU (ms): " << milliseconds << "\n";
}