#include "hip/hip_runtime.h"
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

__global__ void thresholdFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height, int channel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * channel;
        unsigned char pixel = 0;
        for (int c = 0; c < channel; ++c) {
            pixel += srcImage[idx + c] / channel;
        }

        if (pixel > 100) {
            for (int c = 0; c < channel; ++c) {
                dstImage[idx + c] = 255; 
            }
        }
        else {
            for (int c = 0; c < channel; ++c) {
                dstImage[idx + c] = 0;
            }
        }
    }
    
}


extern "C" void thresholdFilter_GPU_wrapper(const cv::Mat & input, cv::Mat & output)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int channel = input.step / input.cols;

    const int inputSize = input.cols * input.rows * channel;
    const int outputSize = output.cols * output.rows * channel;
    unsigned char* d_input, * d_output;

    hipMalloc<unsigned char>(&d_input, inputSize);
    hipMalloc<unsigned char>(&d_output, outputSize);

    hipMemcpy(d_input, input.ptr(), inputSize, hipMemcpyHostToDevice);

    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    hipEventRecord(start);

    thresholdFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, channel);

    hipEventRecord(stop);
    hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    hipEventSynchronize(stop);
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "\nProcessing time for GPU (ms): " << milliseconds << "\n";
}